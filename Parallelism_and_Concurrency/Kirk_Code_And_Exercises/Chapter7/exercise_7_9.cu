#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>
#include <iostream>

double***& generate_matrix( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double*** matrix = new double**[ size ];

	for( auto i = 0; i < size; ++i ){

		matrix[ i ] = new double*[ size ];

		for( auto j = 0; j < size; ++j ){

			matrix[ i ][ j ] = new double[ size ];

			for( int k = 0; k < size; ++k ){

				matrix[ i ][ j ][ k ] = dist( rng );
			}
		}
	}

	return matrix;
}

double*& flatten_matrix( double***& matrix, int size ){

	double* flat = new double[ size * size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				flat[ ( i * size * size ) + ( j * size ) + k ] = matrix[ i ][ j ][ k ];
			}
		}
	}

	return flat;
}

double**& generate_filter( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double** filter = new double*[ size ];

	for( auto i = 0; i < size; ++i ){

		filter[ i ] = new double[ size ];

		for( auto j = 0; j < size; ++j ){

			filter[ i ][ j ] = dist( rng );
		}
	}

	return filter;
}

double*& flatten_filter( double**& filter, int size ){

	double* out = new double[ size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			out[ ( i * size ) + j ] = filter[ i ][ j ];
		}
	}

	return out;
}

void print_matrix( double*** matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ i ][ j ][ k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_matrix( double* matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ ( i * size * size ) + ( j * size ) + k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_filter( double* filter, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			printf( "%f ", filter[ ( i * size ) + j ] );
		}

		printf( "\n" );
	}
}

/// 9. Revise the 2D kernel in Fig. 7.9 to perform 3D convolution.

/// ANSWER: 

#define FILTER_RADIUS 1

__constant__ double filter[ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ];

__global__ void convolution_3D( double* input_matrix, double* output_matrix, int matrix_width, int matrix_height, int matrix_depth  ){
	
	// DESCRIPTION: take the radius R, and pick an element E from the input matrix M, where the indexes of E are x, y, z. Then, all elements within the 3D halo of E in M that share the
	//							same x, y coordinates but differ on z will be multiplied by the same element from the filter. I.e., to get a better visualization, imagine that we are building a 3D ma-
	//							trix with ( 2R + 1 ) slices, where all slices are equal to the filter.

	int outCol = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int outRow = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int outSlice = ( blockIdx.z * blockDim.z ) + threadIdx.z;

	double Pvalue = 0.0f;

	// double filter_element{ 0 }, matrix_element{ 0 }; DEBUG ONLY

	for( auto fSlice = 0; fSlice < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fSlice ){

		for( auto fRow = 0; fRow < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fRow ){

			for( auto fCol = 0; fCol < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fCol ){

				int inRow = outRow - FILTER_RADIUS + fRow;
				int inCol = outCol - FILTER_RADIUS + fCol;
				int inSlice = outSlice - FILTER_RADIUS + fSlice;

				if( inRow >= 0 && inRow < matrix_height &&
					 inCol >= 0 && inCol < matrix_width &&
					 inSlice >= 0 && inSlice < matrix_depth ){

					// filter_element = filter[ ( fRow * ( ( 2 * radius ) + 1 ) ) + fCol ]; DEBUG ONLY
					// matrix_element = input_matrix[ ( inSlice * matrix_depth * matrix_depth ) + ( inRow * matrix_width ) + inCol ]; DEBUG ONLY

					// Here, since our filter is 2D, there is NO need to iterate over its slices, since we are using only one slice repeatedly.
					Pvalue += filter[ fSlice ][ fRow ][ fCol ] * input_matrix[ ( inSlice * matrix_depth * matrix_depth ) + ( inRow * matrix_width ) + inCol ];
				}
			}
		}
	}

	output_matrix[ ( outSlice * matrix_depth * matrix_depth ) + ( outRow * matrix_width ) + outCol ] = Pvalue;
}

// NOTE: both input and output matrices have the SAME dimensions
void set_up( double*& host_InMatrix, double*& host_filter, double*& host_OutMatrix, int host_matrix_height, int host_matrix_width, int host_matrix_depth ){

	double* dev_InMatrix{ nullptr }, *dev_OutMatrix{ nullptr };

	long int matrix_dimensions = host_matrix_depth * host_matrix_width * host_matrix_height * sizeof( double );
	long int filter_dimensions = ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * sizeof( double );

	hipMalloc( ( void** ) &dev_InMatrix, matrix_dimensions );
	hipMalloc( ( void** ) &dev_OutMatrix, matrix_dimensions );

	hipMemcpy( dev_InMatrix, host_InMatrix, matrix_dimensions, hipMemcpyHostToDevice );
	hipMemcpy( dev_OutMatrix, host_OutMatrix, matrix_dimensions, hipMemcpyHostToDevice );
	hipMemcpyToSymbol( HIP_SYMBOL(filter), host_filter, filter_dimensions ); // Copying data into constant memory

	int num_blocks{ 0 }, num_threads{ 0 };

	std::cout << "Enter the amount of blocks and threads: ";
	std::cin >> num_blocks >> num_threads;

	dim3 grid( num_blocks,1, 1 );
	dim3 block( num_threads, num_threads, num_threads );

	//	convolution_3D<<<grid, block>>>( dev_InMatrix, dev_filter, dev_OutMatrix, radius, host_matrix_height, host_matrix_width, host_matrix_depth );
	convolution_3D<<<grid, block>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth );

	hipMemcpy( host_OutMatrix, dev_OutMatrix, matrix_dimensions, hipMemcpyDeviceToHost );

	hipFree( dev_InMatrix );
	hipFree( dev_OutMatrix );
}

int main( ){

	double*** matrix = generate_matrix( 5 );
	double*** filter = generate_matrix( 3 );

	double* flat_matrix = flatten_matrix( matrix, 5 );
	double* flat_filter = flatten_matrix( filter, 3 );

	double* output = new double[ 5 * 5 * 5 * sizeof( double ) ];

	print_matrix( matrix, 5 );

	std::cout << "\n";

	//print_matrix( flat_matrix, 5 );

	//std::cout << "\n";

	print_matrix( flat_filter, 3 );

	std::cout << "\n";

	set_up( flat_matrix, flat_filter, output, 5, 5, 5 );

	std::cout << "\n";

	print_matrix( output, 5 );

	delete[ ] matrix, filter, output;
	delete[ ] flat_matrix, flat_filter;
}