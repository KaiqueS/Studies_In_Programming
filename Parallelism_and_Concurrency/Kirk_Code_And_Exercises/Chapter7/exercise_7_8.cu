#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <iostream>

double***& generate_matrix( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double*** matrix = new double**[ size ];

	for( auto i = 0; i < size; ++i ){

		matrix[ i ] = new double*[ size ];

		for( auto j = 0; j < size; ++j ){

			matrix[ i ][ j ] = new double[ size ];

			for( int k = 0; k < size; ++k ){

				matrix[ i ][ j ][ k ] = dist( rng );
			}
		}
	}

	return matrix;
}

double*& flatten_matrix( double***& matrix, int size ){

	double* flat = new double[ size * size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				flat[ ( i * size * size ) + ( j * size ) + k ] = matrix[ i ][ j ][ k ];
			}
		}
	}

	return flat;
}

void print_matrix( double*** matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ i ][ j ][ k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_matrix( double* matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ ( i * size * size ) + ( j * size ) + k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

/// PROBLEM: 8. Revise the 2D kernel in Fig. 7.7 to perform 3D convolution.
		// NOTE: there are two possible ways of solving that, i.e., using 2D and 3D filters. With 2D filters, we can reapply the filter to radius slices( 3rd dimension of the matrix ) of
		//			 our input matrix. Or, if filters are also 3D, we just assign radius slices of our filter to their respective slices on the input matrix. I think the second approach makes
		//			 more sense than the first, but I will implement both.

/// ANSWER:

// NOTE: both input and output matrices have the SAME dimensions
// Approach: here, I implement the first approach described above, i.e., a 2D filter applied to a 3D matrix.
__global__ void convolution_3D( double* input_matrix, double* filter, double* output_matrix, int radius, int matrix_width, int matrix_height, int matrix_depth  ){
	
	// DESCRIPTION: take the radius R, and pick an element E from the input matrix M, where the indexes of E are x, y, z. Then, all elements within the 3D halo of E in M that share the
	//							same x, y coordinates but differ on z will be multiplied by the same element from the filter. I.e., to get a better visualization, imagine that we are building a 3D ma-
	//							trix with ( 2R + 1 ) slices, where all slices are equal to the filter.

	int outCol = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int outRow = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	int outSlice = ( blockIdx.z * blockDim.z ) + threadIdx.z;

	double Pvalue = 0.0f;

	for( auto fSlice = 0; fSlice < ( ( 2 * radius ) + 1 ); ++fSlice ){

		for( auto fRow = 0; fRow < ( ( 2 * radius ) + 1 ); ++fRow ){

			for( auto fCol = 0; fCol < ( ( 2 * radius ) + 1 ); ++fCol ){

				int inRow = outRow - radius + fRow;
				int inCol = outCol - radius + fCol;
				int inSlice = outSlice - radius + fSlice;

				if( inRow >= 0 && inRow < matrix_height &&
					 inCol >= 0 && inCol < matrix_width &&
					 inSlice >= 0 && inSlice < matrix_depth ){

					Pvalue += filter[ ( fSlice *  ) + (  ) + fCol ] * input_matrix[ ( inSlice * matrix_depth * matrix_depth ) + ( inRow * matrix_width  ) + inCol ];
				}
			}
		}
	}
}

// NOTE: both input and output matrices have the SAME dimensions
void set_up( double*& host_InMatrix, double*& host_filter, double*& host_OutMatrix, int radius, int host_matrix_height, int host_matrix_width, int host_matrix_depth ){

}

int main( ){

	double*** matrix = generate_matrix( 3 );

	double* flat = flatten_matrix( matrix, 3 );

	print_matrix( matrix, 3 );

	std::cout << "\n";

	print_matrix( flat, 3 );

	std::cout << "\n";

	delete[ ] matrix;
	delete[ ] flat;
}