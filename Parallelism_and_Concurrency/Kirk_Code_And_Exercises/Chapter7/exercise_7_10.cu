
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>
#include <iostream>

double***& generate_matrix( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double*** matrix = new double**[ size ];

	for( auto i = 0; i < size; ++i ){

		matrix[ i ] = new double*[ size ];

		for( auto j = 0; j < size; ++j ){

			matrix[ i ][ j ] = new double[ size ];

			for( int k = 0; k < size; ++k ){

				matrix[ i ][ j ][ k ] = dist( rng );
			}
		}
	}

	return matrix;
}

double*& flatten_matrix( double***& matrix, int size ){

	double* flat = new double[ size * size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				flat[ ( i * size * size ) + ( j * size ) + k ] = matrix[ i ][ j ][ k ];
			}
		}
	}

	return flat;
}

double**& generate_filter( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double** filter = new double*[ size ];

	for( auto i = 0; i < size; ++i ){

		filter[ i ] = new double[ size ];

		for( auto j = 0; j < size; ++j ){

			filter[ i ][ j ] = dist( rng );
		}
	}

	return filter;
}

double*& flatten_filter( double**& filter, int size ){

	double* out = new double[ size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			out[ ( i * size ) + j ] = filter[ i ][ j ];
		}
	}

	return out;
}

void print_matrix( double*** matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ i ][ j ][ k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_matrix( double* matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ ( i * size * size ) + ( j * size ) + k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_filter( double* filter, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			printf( "%f ", filter[ ( i * size ) + j ] );
		}

		printf( "\n" );
	}
}

/// 10. Revise the tiled 2D kernel in Fig. 7.12 to perform 3D convolution.

/// ANSWER: 

#define FILTER_RADIUS 1
#define IN_TILE_DIM 16 // I could use extern here instead of a macro, and maybe complicate things, but for now I am avoiding it
#define OUT_TILE_DIM ( ( IN_TILE_DIM ) - ( 2 * ( FILTER_RADIUS ) ) )

__constant__ double filter[ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ];

__global__ void constM_tiled_convolution_3d( double* input_matrix, double* output_matrix, int height, int width, int depth ){

	int slice = ( blockIdx.z * OUT_TILE_DIM ) + threadIdx.z - FILTER_RADIUS;
	int row = ( blockIdx.y * OUT_TILE_DIM ) + threadIdx.y - FILTER_RADIUS;
	int col = ( blockIdx.x * OUT_TILE_DIM ) + threadIdx.x - FILTER_RADIUS;

	__shared__ double shared_input[ IN_TILE_DIM ][ IN_TILE_DIM ][ IN_TILE_DIM ];

	if( ( slice >= 0 && slice < depth ) && ( row >= 0 && row < height ) && ( col >= 0 && col < width ) ){

		shared_input[ threadIdx.z ][ threadIdx.y ][ threadIdx.x ] = input_matrix[ ( slice * depth * depth ) + ( row * height ) + col ];
	}

	else{

		shared_input[ threadIdx.z ][ threadIdx.y ][ threadIdx.x ] = 0.0;
	}

	__syncthreads( );

	int tileSlice = threadIdx.z - FILTER_RADIUS;
	int tileRow = threadIdx.y - FILTER_RADIUS;
	int tileCol = threadIdx.x - FILTER_RADIUS;

	double test{ 0 }, filter_test{ 0 };

	if( ( slice >= 0 && slice < depth ) && ( row >= 0 && row < height ) && ( col >= 0 && col < width ) ){

		if( ( tileSlice >= 0 && tileSlice < OUT_TILE_DIM ) && ( tileRow >= 0 && tileRow < OUT_TILE_DIM ) && ( tileCol >= 0 && tileCol < OUT_TILE_DIM ) ){

			double Pvalue = 0.0;

			for( int fSlice = 0; fSlice < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fSlice ){

				for( int fRow = 0; fRow < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fRow ){

					for( int fCol = 0; fCol < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fCol ){

						test = shared_input[ tileSlice + fSlice ][ tileRow + fRow ][ tileCol + fCol ];
						filter_test = filter[ fSlice ][ fRow ][ fCol ];

						// The problem is here: threads on the block's boundaries are accessing invalid addresses from shared_input
						Pvalue += filter[ fSlice ][ fRow ][ fCol ] * shared_input[ tileSlice + fSlice ][ tileRow + fRow ][ tileCol + fCol ];
					}
				}
			}

			output_matrix[ ( slice * depth * depth ) + ( row * height ) + col ] = Pvalue;
		}
	}
}

// NOTE: both input and output matrices have the SAME dimensions
void set_up( double*& host_InMatrix, double*& host_filter, double*& host_OutMatrix, int host_matrix_height, int host_matrix_width, int host_matrix_depth ){

	double* dev_InMatrix{ nullptr }, *dev_OutMatrix{ nullptr };

	long int matrix_dimensions = host_matrix_depth * host_matrix_width * host_matrix_height * sizeof( double );
	long int filter_dimensions = ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * sizeof( double );

	hipMalloc( ( void** ) &dev_InMatrix, matrix_dimensions );
	hipMalloc( ( void** ) &dev_OutMatrix, matrix_dimensions );

	hipMemcpy( dev_InMatrix, host_InMatrix, matrix_dimensions, hipMemcpyHostToDevice );
	hipMemcpy( dev_OutMatrix, host_OutMatrix, matrix_dimensions, hipMemcpyHostToDevice );
	hipMemcpyToSymbol( HIP_SYMBOL(filter), host_filter, filter_dimensions ); // Copying data into constant memory

	int num_blocks{ 0 }, num_threads{ 0 };

	std::cout << "Enter the amount of blocks and threads: ";
	std::cin >> num_blocks >> num_threads;

	dim3 grid( num_blocks,1, 1 );
	dim3 block( num_threads, num_threads, num_threads );

	//	convolution_3D<<<grid, block>>>( dev_InMatrix, dev_filter, dev_OutMatrix, radius, host_matrix_height, host_matrix_width, host_matrix_depth );
	constM_tiled_convolution_3d<<<grid, block>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth );

	hipMemcpy( host_OutMatrix, dev_OutMatrix, matrix_dimensions, hipMemcpyDeviceToHost );

	hipFree( dev_InMatrix );
	hipFree( dev_OutMatrix );
	hipFree( filter );
}

/// PROBLEM: for the program to run correctly, the block size must be GREATER THAN the input matrix size

int main( ){

	int matrix_dim{ 0 }, filter_dim{ 0 };

	std::cout << "Enter the dimensions of the matrix: ";
	std::cin >> matrix_dim;

	std::cout << "\nEnter the dimensions of the filter matrix: ";
	std::cin >> filter_dim;

	std::cout << "\n";

	double*** matrix = generate_matrix( matrix_dim );
	double*** filter = generate_matrix( filter_dim );

	double* flat_matrix = flatten_matrix( matrix, matrix_dim );
	double* flat_filter = flatten_matrix( filter, filter_dim );

	double* output = new double[ matrix_dim * matrix_dim * matrix_dim * sizeof( double ) ];

	print_matrix( matrix, matrix_dim );

	std::cout << "\n";

	//print_matrix( flat_matrix, 5 );

	//std::cout << "\n";

	print_matrix( flat_filter, filter_dim );

	std::cout << "\n";

	set_up( flat_matrix, flat_filter, output, matrix_dim, matrix_dim, matrix_dim );

	std::cout << "\n";

	print_matrix( output, matrix_dim );

	delete[ ] matrix, filter, output;
	delete[ ] flat_matrix, flat_filter;
}