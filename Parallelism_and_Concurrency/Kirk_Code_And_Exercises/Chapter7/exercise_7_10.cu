#include "hip/hip_runtime.h"


#include <stdio.h>
#include <random>
#include <iostream>

double***& generate_matrix( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double*** matrix = new double**[ size ];

	for( auto i = 0; i < size; ++i ){

		matrix[ i ] = new double*[ size ];

		for( auto j = 0; j < size; ++j ){

			matrix[ i ][ j ] = new double[ size ];

			for( int k = 0; k < size; ++k ){

				matrix[ i ][ j ][ k ] = dist( rng );
			}
		}
	}

	return matrix;
}

double*& flatten_matrix( double***& matrix, int size ){

	double* flat = new double[ size * size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				flat[ ( i * size * size ) + ( j * size ) + k ] = matrix[ i ][ j ][ k ];
			}
		}
	}

	return flat;
}

double**& generate_filter( int size ){

	std::random_device dev;
	std::uniform_real_distribution<double> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	double** filter = new double*[ size ];

	for( auto i = 0; i < size; ++i ){

		filter[ i ] = new double[ size ];

		for( auto j = 0; j < size; ++j ){

			filter[ i ][ j ] = dist( rng );
		}
	}

	return filter;
}

double*& flatten_filter( double**& filter, int size ){

	double* out = new double[ size * size ];

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			out[ ( i * size ) + j ] = filter[ i ][ j ];
		}
	}

	return out;
}

void print_matrix( double*** matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ i ][ j ][ k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_matrix( double* matrix, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			for( auto k = 0; k < size; ++k ){

				printf( "%f ", matrix[ ( i * size * size ) + ( j * size ) + k ] );
			}

			printf( "\n" );
		}

		printf( "\n" );
	}
}

void print_filter( double* filter, int size ){

	for( auto i = 0; i < size; ++i ){

		for( auto j = 0; j < size; ++j ){

			printf( "%f ", filter[ ( i * size ) + j ] );
		}

		printf( "\n" );
	}
}

/// 7.10 - Revise the tiled 2D kernel in Fig. 7.12 to perform 3D convolution.

/// ANSWER: 

#define FILTER_RADIUS 1
#define IN_TILE_DIM 16 // I could use extern here instead of a macro, and maybe complicate things, but for now I am avoiding it
#define OUT_TILE_DIM ( ( IN_TILE_DIM ) - ( 2 * ( FILTER_RADIUS ) ) )

__constant__ double filter[ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ][ ( 2 * FILTER_RADIUS ) + 1 ];

/*__global__ void constM_tiled_convolution_3d( double* input_matrix, double* output_matrix, int height, int width, int depth, int shared_memsize ){

	// Output matrix slice, row, and column
	int slice = ( blockIdx.z * OUT_TILE_DIM ) + threadIdx.z - FILTER_RADIUS;
	int row = ( blockIdx.y * OUT_TILE_DIM ) + threadIdx.y - FILTER_RADIUS;
	int col = ( blockIdx.x * OUT_TILE_DIM ) + threadIdx.x - FILTER_RADIUS;

	//__shared__ double shared_input[ IN_TILE_DIM ][ IN_TILE_DIM ][ IN_TILE_DIM ];
	__shared__ extern double shared_input[];

	if( ( slice >= 0 && slice < depth ) && ( row >= 0 && row < height ) && ( col >= 0 && col < width ) ){

		//shared_input[ threadIdx.z ][ threadIdx.y ][ threadIdx.x ] = input_matrix[ ( slice * depth * depth ) + ( row * height ) + col ];
		shared_input[ (  ) ] = input_matrix[ ( slice * depth * depth ) + ( row * height ) + col ];
	}

	else{

		//shared_input[ threadIdx.z ][ threadIdx.y ][ threadIdx.x ] = 0.0;
		shared_input[  ] = 0.0;
	}

	__syncthreads( );

	int tileSlice = threadIdx.z - FILTER_RADIUS;
	int tileRow = threadIdx.y - FILTER_RADIUS;
	int tileCol = threadIdx.x - FILTER_RADIUS;

	double test{ 0 }, filter_test{ 0 };

	if( ( slice >= 0 && slice < depth ) && ( row >= 0 && row < height ) && ( col >= 0 && col < width ) ){

		if( ( tileSlice >= 0 && tileSlice < OUT_TILE_DIM ) && ( tileRow >= 0 && tileRow < OUT_TILE_DIM ) && ( tileCol >= 0 && tileCol < OUT_TILE_DIM ) ){

			double Pvalue = 0.0;

			for( int fSlice = 0; fSlice < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fSlice ){

				for( int fRow = 0; fRow < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fRow ){

					for( int fCol = 0; fCol < ( ( 2 * FILTER_RADIUS ) + 1 ); ++fCol ){

						test = shared_input[ tileSlice + fSlice ][ tileRow + fRow ][ tileCol + fCol ];
						filter_test = filter[ fSlice ][ fRow ][ fCol ];

						// The problem is here: threads on the block's boundaries are accessing invalid addresses from shared_input
						Pvalue += filter[ fSlice ][ fRow ][ fCol ] * shared_input[ tileSlice + fSlice ][ tileRow + fRow ][ tileCol + fCol ];
					}
				}
			}

			output_matrix[ ( slice * depth * depth ) + ( row * height ) + col ] = Pvalue;
		}
	}
	

}*/

__global__ void constM_tiled_convolution_3d( double* input_matrix, double* output_matrix, int height, int width, int depth, int radius ){

	const int out_tile_dim = blockDim.z - ( 2 * radius );

	// Blocks' dimensions matches input tile + radius BEFORE this function is called
	int in_tile_slice = ( blockDim.z * blockIdx.z ) + threadIdx.z;
	int in_tile_row = ( blockDim.y * blockIdx.y ) + threadIdx.y;
	int in_tile_col = ( blockDim.x * blockIdx.x ) + threadIdx.x;

	// Output matrix is in Global Memory, which means that, if we use multiple blocks, we must use blockDim and blockId to be able to assign threads from different
	// blocks to their corresponding tiles on the output matrix.
	// blockDim - FILTER_RADIUS = output tile dimensions
	// Subtracting FILTER_RADIUS at the end of the expression( after threadIdx ) messes up the following conditional, because output_slice/row/col that should have been greater than 0 are actually equal to it
	int output_slice = ( ( blockDim.z - radius ) * blockIdx.z ) + threadIdx.z - radius;
	int output_row = ( ( blockDim.y - radius ) * blockIdx.y ) + threadIdx.y - radius;
	int output_col = ( ( blockDim.x - radius ) * blockIdx.x ) + threadIdx.x - radius;

	// Shared memory allocation matches block dimensions by definition, i.e., at the call of this function
	__shared__ extern double shared_input[];

	// Loading into shared memory
	if( output_slice >= 0 && output_slice < depth && output_row >= 0 && output_row < height && output_col >= 0 && output_col < width ){

		// By assumption, input and output matrices have the SAME dimensions
		// ATTENTION: shared_input is a TILE, i.e., it is not equal to the entirety of the input matrix.
		//shared_input[ ( in_tile_slice * ( blockDim.z * blockDim.z ) ) + ( in_tile_row * ( blockDim.y - FILTER_RADIUS ) ) + ( blockDim.x - FILTER_RADIUS ) ] = input_matrix[ ( output_slice * depth * depth ) + ( output_row * height ) + output_col ];
		shared_input[ ( threadIdx.z * ( blockDim.z * blockDim.z ) ) + ( threadIdx.y * blockDim.y ) + threadIdx.x ] = input_matrix[ ( output_slice * depth * depth ) + ( output_row * height ) + output_col ];
	}

	else{

		//shared_input[ ( in_tile_slice * ( blockDim.z * blockDim.z ) ) + ( in_tile_row * ( blockDim.y - FILTER_RADIUS ) ) + ( blockDim.x - FILTER_RADIUS ) ] = 0.0;
		shared_input[ ( threadIdx.z * ( blockDim.z * blockDim.z ) ) + ( threadIdx.y * blockDim.y ) + threadIdx.x ] = 0.0;
	}

	__syncthreads( );

	// PROBLEM: this should not be working when ( blockdim - radius ) < input/output_matrix size
		// POTENTIAL ANSWER: since shared_input > input/output matrices dimensions, there is always a subset of threads within valid ranges
		//                   Also, by removing num_threads += radius, things got fixed. I say fuck it.
	/*if( output_slice >= 0 && output_slice < depth && output_row >= 0 && output_row < height && output_col >= 0 && output_col < width ){

		//output_matrix[ ( output_slice * depth * depth ) + ( output_row * height ) + output_col ] = shared_input[ ( in_tile_slice * ( blockDim.z * blockDim.z ) ) + ( in_tile_row * ( blockDim.y - FILTER_RADIUS ) ) + ( blockDim.x - FILTER_RADIUS ) ];
		output_matrix[ ( output_slice * depth * depth ) + ( output_row * height ) + output_col ] = shared_input[ ( threadIdx.z * ( blockDim.z * blockDim.z ) ) + ( threadIdx.y * blockDim.y ) + threadIdx.x ];
	}*/
	
	// NOTE: I still have to explain myself this and why it is necessary to subtract radius from threadId
	int intile_slice = threadIdx.z - radius;
	int intile_row = threadIdx.y - radius;
	int intile_col = threadIdx.x - radius;

	if( output_slice >= 0 && output_slice < depth && output_row >= 0 && output_row < height && output_col >= 0 && output_col < width ){

		if( intile_slice >= 0 && intile_slice < ( blockDim.z - radius ) && intile_row >= 0 && intile_row < ( blockDim.y - radius ) && intile_col >= 0 && intile_col < ( blockDim.x - radius ) ){

			double Pvalue{ 0.0 };

			for( int fSlice = 0; fSlice < ( 2 * radius ) + 1; ++fSlice ){

				for( int fRow = 0; fRow < ( 2 * radius ) + 1; ++fRow ){

					for( int fCol = 0; fCol < ( 2 * radius ) + 1; ++fCol ){

						Pvalue += filter[ fSlice ][ fRow ][ fCol ] * shared_input[ ( ( intile_slice + fSlice ) * ( blockDim.z * blockDim.z ) ) + ( ( intile_row + fRow ) * blockDim.y ) + ( intile_col + fCol ) ];
					}
				}
			}

			output_matrix[ ( output_slice * depth * depth ) + ( output_row * height ) + output_col ] += Pvalue;
		}
	}
}

// NOTE: both input and output matrices have the SAME dimensions
/*void set_up( double*& host_InMatrix, double*& host_filter, double*& host_OutMatrix, int host_matrix_height, int host_matrix_width, int host_matrix_depth ){

	double* dev_InMatrix{ nullptr }, *dev_OutMatrix{ nullptr };

	long int matrix_dimensions = host_matrix_depth * host_matrix_width * host_matrix_height * sizeof( double );
	long int filter_dimensions = ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * ( ( 2 * FILTER_RADIUS ) + 1 ) * sizeof( double );

	cudaMalloc( ( void** ) &dev_InMatrix, matrix_dimensions );
	cudaMalloc( ( void** ) &dev_OutMatrix, matrix_dimensions );

	cudaMemcpy( dev_InMatrix, host_InMatrix, matrix_dimensions, cudaMemcpyHostToDevice );
	cudaMemcpy( dev_OutMatrix, host_OutMatrix, matrix_dimensions, cudaMemcpyHostToDevice );
	cudaMemcpyToSymbol( filter, host_filter, filter_dimensions ); // Copying data into constant memory

	int num_blocks{ 0 }, num_threads{ 0 };

	std::cout << "Enter the amount of blocks and threads: ";
	std::cin >> num_blocks >> num_threads;

	dim3 grid( num_blocks,1, 1 );
	dim3 block( num_threads, num_threads, num_threads );

	//int shared_memsize{ 0 };

	//std::cout << "\nEnter the Shared Memory size: ";
	//std::cin >> shared_memsize;

	//shared_memsize = ( shared_memsize * shared_memsize * shared_memsize ) * sizeof( double );

	//convolution_3D<<<grid, block>>>( dev_InMatrix, dev_filter, dev_OutMatrix, radius, host_matrix_height, host_matrix_width, host_matrix_depth );
	//constM_tiled_convolution_3d<<<grid, block, shared_memsize>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth );
	constM_tiled_convolution_3d<<<grid, block>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth );

	cudaMemcpy( host_OutMatrix, dev_OutMatrix, matrix_dimensions, cudaMemcpyDeviceToHost );

	cudaFree( dev_InMatrix );
	cudaFree( dev_OutMatrix );
	cudaFree( filter );
}*/

void set_up( double*& host_InMatrix, double*& host_filter, double*& host_OutMatrix, int host_matrix_height, int host_matrix_width, int host_matrix_depth ){

	int radius{ 1 };

	double* dev_InMatrix{ nullptr }, *dev_OutMatrix{ nullptr };

	long int matrix_dimensions = host_matrix_depth * host_matrix_width * host_matrix_height * sizeof( double );
	long int filter_dimensions = ( ( 2 * radius ) + 1 ) * ( ( 2 * radius ) + 1 ) * ( ( 2 * radius ) + 1 ) * sizeof( double );

	hipMalloc( ( void** ) &dev_InMatrix, matrix_dimensions );
	hipMalloc( ( void** ) &dev_OutMatrix, matrix_dimensions );

	hipMemcpy( dev_InMatrix, host_InMatrix, matrix_dimensions, hipMemcpyHostToDevice );
	//cudaMemcpy( dev_OutMatrix, host_OutMatrix, matrix_dimensions, cudaMemcpyHostToDevice );
	hipMemcpyToSymbol( HIP_SYMBOL(filter), host_filter, filter_dimensions ); // Copying data into constant memory

	int num_blocks{ 0 }, num_threads{ 0 };

	std::cout << "Enter the amount of blocks and threads: ";
	std::cin >> num_blocks >> num_threads;

	num_threads += radius;

	dim3 grid( num_blocks, 1, 1 );
	dim3 block( num_threads, num_threads, num_threads );

	int shared_memsize = ( num_threads * num_threads * num_threads ) * sizeof( double );

	//std::cout << "\nEnter the Shared Memory size: ";
	//std::cin >> shared_memsize;

	//shared_memsize = ( shared_memsize * shared_memsize * shared_memsize ) * sizeof( double );

	//convolution_3D<<<grid, block>>>( dev_InMatrix, dev_filter, dev_OutMatrix, radius, host_matrix_height, host_matrix_width, host_matrix_depth );
	//constM_tiled_convolution_3d<<<grid, block, shared_memsize>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth );
	constM_tiled_convolution_3d<<<grid, block, shared_memsize>>>( dev_InMatrix, dev_OutMatrix, host_matrix_height, host_matrix_width, host_matrix_depth, radius );

	hipMemcpy( host_OutMatrix, dev_OutMatrix, matrix_dimensions, hipMemcpyDeviceToHost );

	hipFree( dev_InMatrix );
	hipFree( dev_OutMatrix );
	hipFree( filter );
}

/// PROBLEM: for the program to run correctly, the block size must be GREATER THAN the input matrix size

int main( ){

	int matrix_dim{ 0 }, filter_dim{ 0 };

	std::cout << "Enter the dimensions of the matrix: ";
	std::cin >> matrix_dim;

	std::cout << "\nEnter the dimensions of the filter matrix: ";
	std::cin >> filter_dim;

	std::cout << "\n";

	double*** matrix = generate_matrix( matrix_dim );
	double*** filter = generate_matrix( filter_dim );

	double* flat_matrix = flatten_matrix( matrix, matrix_dim );
	double* flat_filter = flatten_matrix( filter, filter_dim );

	double* output = new double[ matrix_dim * matrix_dim * matrix_dim * sizeof( double ) ];

	print_matrix( matrix, matrix_dim );

	std::cout << "\n";

	//print_matrix( flat_matrix, 5 );

	//std::cout << "\n";

	print_matrix( flat_filter, filter_dim );

	std::cout << "\n";

	set_up( flat_matrix, flat_filter, output, matrix_dim, matrix_dim, matrix_dim );

	std::cout << "\n";

	print_matrix( output, matrix_dim );

	delete[ ] matrix, filter, output;
	delete[ ] flat_matrix, flat_filter;
}