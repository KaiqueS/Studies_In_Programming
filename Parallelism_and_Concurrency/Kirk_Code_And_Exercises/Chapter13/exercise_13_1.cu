#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <random>
#include <bitset>

unsigned int*& generate_array( int size ) {

	unsigned int* array = new unsigned int[ size ];

	std::random_device dev;
	std::uniform_int_distribution<int> dist( -( size * size ), ( size * size ) );
	std::mt19937_64 rng( dev( ) );

	for( auto i = 0; i < size; ++i ) {

		array[ i ] = dist( rng );
	}

	return array;
}

void print( unsigned int*& array, int size ) {

	for( auto i = 0; i < size; ++i ) {

		std::cout << std::bitset<32>( array[ i ] ) << " ";
	}

	printf( "\n" );
}

/// PROBLEM: Extend the kernel in Fig. 13.4 by using shared memory to improve memory coalescing.

/// ANSWER:

// NOTE/PROBLEM: shared memory, even in nested kernel calls, does not persist through calls. Thus, passing a shared memory array as argument to the
//				 unsigned int* bits parameters results in bits, within exclusiveScan, being different from the shared memory array that initializes
//				 it. I.e., bits, within exclusiveScan, != its argument.
__global__ void exclusiveScan( unsigned int* bits, unsigned int* output, unsigned int N, int* flags, int* scan_value, int blockCounter ) {

	// NOTE: potential cause of the problem
	__shared__ unsigned int bid_s;

	if( threadIdx.x == 0 ) {

		bid_s = atomicAdd( &blockCounter, 1 );
	}

	__syncthreads( );

	unsigned int bid = bid_s;

	// NOTE: potential cause of the problem
	__shared__ extern unsigned int SharedMem[ ];

	unsigned int i = ( blockDim.x * blockIdx.x ) + threadIdx.x;

	// NOTE: since we are adding up bits, we default to 0 if the following condition is not met, because
	//		 0 is the addition identity
	SharedMem[ threadIdx.x ] = ( ( i < N ) && ( threadIdx.x != 0 ) ) ? bits[ i - 1 ] : 0;

	// NOTE: what the hell is this adding up? We should not add up bits, we should count them.
				// No, we are actually adding them.
	for( unsigned int stride = 1; stride < blockDim.x; stride *= 2 ){

		__syncthreads( );

		unsigned int temp{ };

		if( threadIdx.x >= stride ){

			temp = SharedMem[ threadIdx.x ] + SharedMem[ threadIdx.x - stride ];
		}

		__syncthreads( );

		if( threadIdx.x >= stride ){

			SharedMem[ threadIdx.x ] = temp;
		}
	}

	if( i < N ){

		// NOTE: substituted output[] for bits[]
		// PROBLEM: data-race. If thread 1 modifies bits before thread 2, shit happens
		//			may atomic add?
		bits[ i ] = SharedMem[ threadIdx.x ];
	}

	// Block Synchronization

	// NOTE: potential cause of the problem
	__shared__ unsigned int previous_sum;

	if( threadIdx.x == 0 ) {

		// bid = blockId
		while( atomicAdd( &flags[ bid ], 0 ) == 0 ) { }

		previous_sum = scan_value[ bid ];

		// NOTE: substituted output[] for bits[]
		scan_value[ bid + 1 ] = previous_sum + bits[ i + blockDim.x ];

		__threadfence( );

		atomicAdd( &flags[ bid + 1 ], 1 );
	}

	__syncthreads( );
}

__global__ void radix_sort_iter( unsigned int* input, unsigned int* output, unsigned int* bits, unsigned int N, unsigned int iter, int* flags, int* scan_value, int blockCounter ) {

	unsigned int i = ( blockIdx.x * blockDim.x ) + threadIdx.x;

	unsigned int key{ 0 }, bit{ 0 };

	if( i < N ) {

		key = input[ i ];
		bit = ( key >> iter ) & 1;
		bits[ i ] = bit;

		//printf( "%d ", bits[ i ] );
	}

	__syncthreads( );

	// Counts the amount of 1's before i
	// PROBLEM: exclusiveScan is NOT counting the amount of 0/1 before the ith position
	exclusiveScan<<<gridDim.x, blockDim.x, blockDim.x * sizeof( unsigned int )>>>( bits, output, N, flags, scan_value, blockCounter );

	if( i < N ) {

		unsigned int OnesBefore = bits[ i ];
		unsigned int OnesTotal = bits[ N - 1 ]; // This means that bits MUST have N elements
		unsigned int dst = ( bit == 0 ) ? ( i - OnesBefore ) : ( N - OnesTotal - OnesBefore );

		//printf( "%d %d ", OnesBefore, OnesTotal );

		output[ dst ] = key;
	}
}

void kernel_setup( unsigned int* host_input, unsigned int* host_output, unsigned int* host_bits, unsigned int host_N ) {

	unsigned int* dev_input{ nullptr }, *dev_output{ nullptr }, *dev_bits{ nullptr };
	
	int* flags{ nullptr }, *dev_flags{ nullptr };
	int* scan_value{ nullptr };

	// NOTE: block_counter is NOT AN ARRAY
	int block_counter{ 0 };
	
	unsigned int array_size = host_N * sizeof( unsigned int );

	hipMalloc( ( void** ) &dev_input, array_size );
	hipMalloc( ( void** ) &dev_output, array_size );
	hipMalloc( ( void** ) &dev_bits, array_size ); // ATTENTION: potentially incorrect size

	hipMemcpy( dev_input, host_input, array_size, hipMemcpyHostToDevice );

	unsigned int num_blocks{ 0 };
	unsigned int num_threads{ 0 };

	std::cout << "\nEnter the number of blocks: ";
	std::cin >> num_blocks;

	hipMalloc( ( void** ) &scan_value, num_blocks * sizeof( int ) );

	std::cout << "\nEnter the number of threads: ";
	std::cin >> num_threads;

	hipMalloc( ( void** ) &dev_flags, num_blocks * sizeof( int ) );

	// NOTE: flags used to indicate which block to run next. If flags[ i ] != 0, for any i, it is i's turn to run.
	//		 After i's turn, the block i sets flags[ i + 1 ] to 1, so block i + 1 can run.
	flags = new int[ num_blocks ];

	flags[ 0 ] = 1;

	for( auto i = 1; i < num_blocks; ++i ) {

		flags[ i ] = 0;
	}

	hipMemcpy( dev_flags, flags, num_blocks * sizeof( int ), hipMemcpyHostToDevice );

	dim3 blocks{ num_blocks };
	dim3 threads{ num_threads };

	unsigned int shared_memsize{ 0 };

	//std::cout << "\nEnter the size of shared memory: ";
	//std::cin >> shared_memsize;

	shared_memsize = host_N * sizeof( unsigned int );

	for( auto iter = 0; iter < ( 8 * sizeof( unsigned int ) ); ++iter ){

		// NOTE: potential cause of the problem shared_memsize might be incorrect
		radix_sort_iter<<<blocks, threads, shared_memsize>>>( dev_input, dev_output, dev_bits, host_N, iter, dev_flags, scan_value, block_counter );
	}

	hipMemcpy( host_output, dev_output, array_size, hipMemcpyDeviceToHost );

	hipFree( dev_input );
	hipFree( dev_output );
	hipFree( dev_bits );
	hipFree( dev_flags );
	hipFree( scan_value );

	delete[ ] flags;
}

// NOTE: Parallel Radix Sort requires calling the sorting kernel ITERATEDLY, where the amount of iterations ranges from 0 to the size in bit of each input
int main( ) {

	int size{ 0 };

	std::cout << "Enter the size of the array: ";
	std::cin >> size;

	unsigned int* array = generate_array( size );
	unsigned int* output = new unsigned int[ size ];
	unsigned int* bits = new unsigned int[ size ];

	print( array, size );

	kernel_setup( array, output, bits, size );

	std::cout << "\n";

	print( output, size );

	delete[ ] array, output, bits;
}