#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <random>

void fill_matrix( float**& matrix, int size ){

    std::random_device dev;
    std::uniform_real_distribution<float> dist( -( size * size ), ( size * size ) );
    std::mt19937_64 rng( dev( ) );

    //int dim = size * size;

    matrix = new float*[ size ];

    for( auto i = 0; i < size; ++i ){

        matrix[ i ] = new float( size );

        for( auto j = 0; j < size; ++j ){

            matrix[ i ][ j ] = dist( rng );
        }
    }
}

void print_matrix( float**& matrix, int size ){

    for( auto i = 0; i < size; ++i ){

        for( auto j = 0; j < size; ++j ){

            printf( "%lf ", matrix[ i ][ j ] );

            if( j == size - 1 ){

                printf( "\n" );
            }
        }
    }
}

void print_matrix( float*& matrix, int size ){

    int double_dim = ( size * size );

    for( auto i = 0; i < double_dim; ++i ){

        printf( "%lf", matrix[ i ] );

        if( double_dim % size == 0 ){

            printf( "\n" );
        }
    }
}

/// 1. A matrix addition takes two input matrices A and B and produces one output matrix C. Each element of the output matrix C is the sum of the corresponding
/// elements of the input matrices A and B, i.e., C[i][j] = A[i][j] + B[i][j]. For simplicity, we will only handle square matrices whose elements are
/// single-precision floating-point numbers. Write a matrix addition kernel and the host stub function that can be called with four parameters: pointer-
/// to-the-output matrix, pointer-to-the-first-input matrix, pointer-to-the-second-input matrix, and the number of elements in each dimension. Follow the
/// instructions below:

/// B. Write a kernel that has each thread to produce one output matrix element. Fill in the execution configuration parameters for this design.

// Problem: how to generalize for instances where the dimensions of the matrix are greater than the available amount of threads?
__global__
void matrixAddKernel_B( float* output, float* first_input, float* second_input, int size ){

    int matrix_range{ size * size };

    int row = ( blockIdx.x * blockDim.x ) + threadIdx.x;
    int col = ( blockIdx.y * blockDim.y ) + threadIdx.y;

    if( ( row < matrix_range ) && ( col < matrix_range ) ){

        output[ ( row * size ) + col ] = first_input[ ( row * size ) + col ] + second_input[ ( row * size ) + col ];
    }
}

// C. Write a kernel that has each thread to produce one output matrix row. Fill in the execution configuration parameters for the design.

__global__
void matrixAddKernel_C( float* output, float* first_input, float* second_input, int size ){

    
}

// D. Write a kernel that has each thread to produce one output matrix column. Fill in the execution configuration parameters for the design.

__global__
void matrixAddKernel_D( float* output, float* first_input, float* second_input, int size ){

    
}

// E. Analyze the pros and cons of each kernel design above.

// A. Write the host stub function by allocating memory for the input and output matrices, transferring input data to device; launch the kernel, transferring the
// output data to host and freeing the device memory for the input and output data. Leave the execution configuration parameters open for this step.

void set_up( float* h_output, float* h_first_input, float* h_second_input, int dim ){

    float *d_Output{ nullptr }, *d_first{ nullptr }, *d_second{ nullptr };
    
    int size = dim * sizeof( float );

    hipMalloc( ( void** ) &d_first, ( size * size ) );
    hipMalloc( ( void** ) &d_second, ( size * size ) );
    hipMalloc( ( void** ) &d_Output, ( size * size ) );

    hipMemcpy( d_first, h_first_input, size, hipMemcpyHostToDevice );
    hipMemcpy( d_second, h_second_input, size, hipMemcpyHostToDevice );

    int blocks{ 0 }, threads{ 0 };

    printf( "Enter the Grid and Block sizes: \n" );
    std::cin >> blocks >> threads;

    dim3 dimGrid( blocks, 1, 1 );
    dim3 dimBlock( threads, 1, 1 );

    matrixAddKernel_B<<<dimGrid, dimBlock>>>( d_Output, d_first, d_second, dim ); // 32,32 are just dummy inputs, else the code would not compile.

    hipMemcpy( h_output, d_Output, size, hipMemcpyDeviceToHost );

    print_matrix( h_output, dim );

    hipFree( d_Output );
    hipFree( d_first );
    hipFree( d_second );
}

int main( ){

    float **output{ nullptr }, **first{ nullptr }, **second{ nullptr };

    int dimension{ 0 };

    std::cout << "Enter the matrix dimensions:" << "\n";
    std::cin >> dimension;

    fill_matrix( first, dimension );
    fill_matrix( second, dimension );

    set_up( *output, *first, *second, dimension );

    std::cout << "\n";

    delete[ ] first, second;
}